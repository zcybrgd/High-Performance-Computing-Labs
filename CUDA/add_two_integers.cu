#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void add ( int *a , int*b , int *c ) {
*c = *a + *b ;
}
int main ( void ) {
int a , b , c ; //copies de a, b, c de Host
int *dev_a , *dev_b , *dev_c ; //pointeurs vers des zones dans le GPU (device)
int size = sizeof ( int ) ;
// allocation de l’espace pour le device
hipMalloc( (void **)&dev_a, size) ;
hipMalloc( (void **)&dev_b, size) ;
hipMalloc( (void **)&dev_c, size) ;
a=2 ;
b=7 ;
// Copie des données vers le Device
hipMemcpy( dev_a , &a , size , hipMemcpyHostToDevice );
hipMemcpy( dev_b , &b , size , hipMemcpyHostToDevice);
add <<< 1 , 1 >>> ( dev_a , dev_b , dev_c ) ;
//Copie du resultat vers Host
hipMemcpy (&c, dev_c, size, hipMemcpyDeviceToHost) ;
//Liberation de l’espace alloué
hipFree (dev_a) ;
hipFree ( dev_b) ;
hipFree ( dev_c) ;
return 0
}


/*
nvcc -o add_two_integers add_two_integers.cu
./add_two_integers

*/
