#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>


#define N 512

__global__ void add ( int *a , int *b , int * c ) {
c [ blockIdx.x ] = a [ blockIdx.x ] + b [ blockIdx.x ] ;
}

int main( void ) {
int *a, *b, *c; //copies de a, b, c de Host
int
*dev_a, *dev_b, *dev_c; //copies de a, b, c de Device
int size = N * sizeof( int);
// allocation de l’espace pour le device
hipMalloc( (void**)&dev_a, size);
hipMalloc( (void**)&dev_b, size);
hipMalloc( (void**)&dev_c, size);

a = (int*)malloc( size );
b = (int*)malloc( size );
c = (int*)malloc( size );
random_ints( a, N );
random_ints( b, N );

// Copie des données vers le Device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);

// Lancer kernel add() kernel avec N parallèles blocs
add<<< N, 1 >>>( dev_a, dev_b, devc);

//Copie du resultat vers Host
hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);

//Liberation de l’espace alloué
free( a ); free( b ); free( c):
hipFree( dev_a );
hipFree( dev_b );
hipFree( dev_c );
return 0; }
