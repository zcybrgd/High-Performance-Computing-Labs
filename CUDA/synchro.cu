#include "hip/hip_runtime.h"
#define N 512

__global__ void dot(int *a, int *b, int *c) {
    __shared__ int temp[N]; // mémoire partagée

    // chaque thread calcule un produit partiel
    temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

    __syncthreads(); // on attend que tous les threads aient fini

    // seul le thread 0 fait la somme
    if (threadIdx.x == 0) {
        int sum = 0;
        for (int i = 0; i < N; i++)
            sum += temp[i];
        *c = sum;
    }
}



int main(void) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, sizeof(int));

    a = (int*)malloc(size);
    b = (int*)malloc(size);

    random_ints(a, N);
    random_ints(b, N);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    dot<<<1, N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    // Nettoyage
    free(a); free(b); free(c);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    return 0;
}
